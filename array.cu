int array_size = 800;
for (i = 0; i < array_size; i++) {
    int t = i + stride;
    if (t >= array_size) t %= stride;
    host_array[i] = (int)device_array + 4*t;
    }
hipMemcpy(device_array, host_array, array_size, hipMemcpyDeviceToHost);